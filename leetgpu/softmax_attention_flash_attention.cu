#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>
#include <limits>
#include <iostream>
#include <stdio.h>
#include <cmath>
#include <algorithm>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

#define CEIL_DIV(dividend, divisor) (((dividend) + (divisor) - 1) / (divisor))

__device__ static inline float onlineSoftmaxSum(float const maxA,
                                                float const sumA,
                                                float const maxB,
                                                float const sumB) {
    if (sumA == 0.0f) {
        return sumB;
    } else if (sumB == 0.0f) {
        return sumA;
    } else if (maxA > maxB) {
        return sumB * expf(maxB - maxA) + sumA;
    } else {
        return sumB + sumA * expf(maxA - maxB);
    }
}

__global__ void flash_attention_kernel(float const* const Q_HBM,  // size Mxd
                                       float const* const K_HBM,  // size Nxd
                                       float const* const V_HBM,  // size Nxd
                                       float* const O_HBM,        // size Mxd
                                       int const M,
                                       int const N,
                                       int const d,
                                       float const temperature,
                                       float* const row_sum_HBM,
                                       float* const row_max_HBM,
                                       int const maxSharedMemory) {
    extern __shared__ float sharedMemory[];
    int const B_c = min(CEIL_DIV(maxSharedMemory, 4 * d * sizeof(float)), (unsigned long)N);
    int const B_r = min(CEIL_DIV(maxSharedMemory, 4 * d * sizeof(float)), (unsigned long)d);
    int const T_c = CEIL_DIV(N, B_c);

    int const B_r_bounds_checked_for_last_row = min(B_r, M - blockIdx.x * B_r);

    float* const Q = sharedMemory;
    float* const K = Q + B_r * d;
    float* const V = K + B_c * d;
    float* const S = V + B_c * d;

    // Initialize S, using threadIdx.x as the B_c dimension.
    for (int B_r_index = 0; B_r_index < B_r; B_r_index++) {
        S[B_r_index * B_c + threadIdx.x] = 0.0f;
    }

    // Load Q, using threadIdx.x to help along the d dimension
    for (int d_index = threadIdx.x; d_index < d; d_index += blockDim.x) {
        for (int B_r_index = 0; B_r_index < B_r_bounds_checked_for_last_row; B_r_index++) {
            int const row_index = blockIdx.x * B_r + B_r_index;
            Q[B_r_index * d + d_index] = Q_HBM[row_index * d + d_index];
        }
    }

    // Iterate horizontally through different S blocks.
    for (int T_c_index = 0; T_c_index < T_c; T_c_index++) {
        int const B_c_bounds_checked_for_last_column = min(B_c, N - T_c_index * B_c);
        // Load K and V
        for (int d_index = threadIdx.x; d_index < d; d_index += blockDim.x) {
            for (int B_c_index = 0; B_c_index < B_c_bounds_checked_for_last_column; B_c_index++) {
                int const row_index = T_c_index * B_c + B_c_index;
                K[B_c_index * d + d_index] = K_HBM[row_index * d + d_index];
                V[B_c_index * d + d_index] = V_HBM[row_index * d + d_index];
            }
        }

        __syncthreads();

        // Iterate vertically within the S block.
        for (int B_r_index = 0; B_r_index < B_r_bounds_checked_for_last_row; B_r_index++) {
            float S_val_for_thread = 0.0f;
            for (int d_index = 0; d_index < d; d_index++) {
                S_val_for_thread += Q[B_r_index * d + d_index] * K[threadIdx.x * d + d_index];
            }
            S[B_r_index * B_c + threadIdx.x] = S_val_for_thread / temperature;

            int const row_index = blockIdx.x * B_r + B_r_index;
            float const S_row_old_global_max = row_max_HBM[row_index];
            float const S_row_old_global_sum = row_sum_HBM[row_index];
            __syncthreads();

            // Update max and sum for this row.
            if (threadIdx.x == 0) {
                float S_row_local_max = -INFINITY;
                float S_row_local_sum = 0.0f;
                for (int col = 0; col < B_c_bounds_checked_for_last_column; col++) {
                    float const S_val_iter = S[B_r_index * B_c + col];
                    S_row_local_sum = onlineSoftmaxSum(S_row_local_max, S_row_local_sum, S_val_iter, 1.0f);
                    S_row_local_max = max(S_row_local_max, S_val_iter);
                }
                row_sum_HBM[row_index] = onlineSoftmaxSum(S_row_old_global_max,
                                                            S_row_old_global_sum,
                                                            S_row_local_max,
                                                            S_row_local_sum);
                row_max_HBM[row_index] = max(S_row_old_global_max, S_row_local_max);
            }
            __syncthreads();
            float const S_row_new_global_max = row_max_HBM[row_index];
            float const S_row_new_global_sum = row_sum_HBM[row_index];

            // Compute P and O
            for (int d_index = threadIdx.x; d_index < d; d_index += blockDim.x) {
                float PV_val = 0.0f;
                for (int V_B_c_index = 0; V_B_c_index < B_c_bounds_checked_for_last_column; V_B_c_index++) {
                    float const S_val = S[B_r_index * B_c + V_B_c_index];
                    float const P_val = expf(S_val - S_row_new_global_max) / S_row_new_global_sum;
                    PV_val += P_val * V[V_B_c_index * d + d_index];
                }

                int const row_index = blockIdx.x * B_r + B_r_index;
                int const OIndexForThread = row_index * d + d_index;
                O_HBM[OIndexForThread] = O_HBM[OIndexForThread] * expf(S_row_old_global_max - S_row_new_global_max) * (S_row_old_global_sum / S_row_new_global_sum) + PV_val;
            }
        }
    }
}


// Q, K, V, output are device pointers
void solve(float const* const Q,  // size Mxd
           float const* const K,  // size Nxd
           float const* const V,  // size Nxd
           float* const output,   // size Mxd
           int const M,
           int const N,
           int const d) {
    int maxSharedMemory;
    gpuErrchk(hipDeviceGetAttribute(&maxSharedMemory, hipDeviceAttributeMaxSharedMemoryPerBlock, 0));
    gpuErrchk(hipFuncSetAttribute(reinterpret_cast<const void*>(flash_attention_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, maxSharedMemory));

    int const B_c = min(CEIL_DIV(maxSharedMemory, 4 * d * sizeof(float)), (unsigned long)N);
    int const B_r = min(CEIL_DIV(maxSharedMemory, 4 * d * sizeof(float)), (unsigned long)d);
    int const T_r = CEIL_DIV(M, B_r);

    std::cout << "maxSharedMemory: " << maxSharedMemory << std::endl;
    std::cout << "B_c: " << B_c << std::endl;
    std::cout << "B_r: " << B_r << std::endl;
    std::cout << "T_r: " << T_r << std::endl;

    float* row_sum_HBM;
    gpuErrchk(hipMalloc((void**)&row_sum_HBM, M * sizeof(float)));
    float* row_max_HBM;
    gpuErrchk(hipMalloc((void**)&row_max_HBM, M * sizeof(float)));

    float* zeroFloats = new float[M*d]();
    gpuErrchk(hipMemcpy(output, zeroFloats, M * d * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(row_sum_HBM, zeroFloats, M * sizeof(float), hipMemcpyHostToDevice));

    float* negativeInfinityFloats = new float[M];
    std::fill(negativeInfinityFloats, negativeInfinityFloats + M, -INFINITY);
    gpuErrchk(hipMemcpy(row_max_HBM, negativeInfinityFloats, M * sizeof(float), hipMemcpyHostToDevice));

    float const temperature = sqrt(d);

    dim3 const blocksPerGrid(T_r);
    dim3 const threadsPerBlock(B_c);
    flash_attention_kernel<<<blocksPerGrid, threadsPerBlock, maxSharedMemory>>>(Q, K, V, output, M, N, d, temperature, row_sum_HBM, row_max_HBM, maxSharedMemory);
    gpuErrchk(hipPeekAtLastError());

    delete[] zeroFloats;
    delete[] negativeInfinityFloats;
}



// Host utility to print Mxd matrix
void print_matrix(const float* mat, int M, int d) {
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < d; ++j)
            std::cout << mat[i * d + j] << " ";
        std::cout << std::endl;
    }
}

int main() {
    const int M = 2000, N = 3000, d = 1024;

    // Input data
    float* const Q_host = new float[M*d]();
    float* const K_host = new float[N*d]();
    float* const V_host = new float[N*d]();

    const size_t size_Q = M * d * sizeof(float);
    const size_t size_KV = N * d * sizeof(float);
    const size_t size_O = size_Q;

    float* Q_dev;
    float* K_dev;
    float* V_dev;
    float* O_dev;

    hipMalloc(&Q_dev, size_Q);
    hipMalloc(&K_dev, size_KV);
    hipMalloc(&V_dev, size_KV);
    hipMalloc(&O_dev, size_O);

    hipMemcpy(Q_dev, Q_host, size_Q, hipMemcpyHostToDevice);
    hipMemcpy(K_dev, K_host, size_KV, hipMemcpyHostToDevice);
    hipMemcpy(V_dev, V_host, size_KV, hipMemcpyHostToDevice);

    solve(Q_dev, K_dev, V_dev, O_dev, M, N, d);
    hipDeviceSynchronize();

    // Copy result back
    float* const O_host = new float[M * d]();
    hipMemcpy(O_host, O_dev, size_O, hipMemcpyDeviceToHost);

    // Print output
    std::cout << "Output O:" << std::endl;
    print_matrix(O_host, M, d);

    // Free memory
    hipFree(Q_dev);
    hipFree(K_dev);
    hipFree(V_dev);
    hipFree(O_dev);

    return 0;
}


