#include "hip/hip_runtime.h"
#include "solve.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define CEIL_DIV(dividend, divisor) (((dividend) + (divisor) - 1) / (divisor))


unsigned int const maxInputsPerSolveCall = 2048;

__global__ void sum_kernel(float const* const input,
                           float* const output,
                           int const N,
                           float* const buffer) {
    // X is the offset into buffer, not into input.
    int const x = blockIdx.x * blockDim.x + threadIdx.x;
    int const bufferLength = (N + 1) / 2;

    if (x < bufferLength) {
        buffer[x] = input[2 * x] + (2 * x + 1 < N ? input[2 * x + 1] : 0);
    }
    __syncthreads();

    for (int distanceFromNeighborToSum = 1;
         distanceFromNeighborToSum < bufferLength;
         distanceFromNeighborToSum *= 2) {
        if (x % (2 * distanceFromNeighborToSum) == 0) {
            buffer[x] = buffer[x] + (x + distanceFromNeighborToSum < bufferLength ? buffer[x + distanceFromNeighborToSum] : 0);
        }
        __syncthreads();
    }
    if (x == 0) {
        *output = buffer[0];
    }
}

// input, output are device pointers
void solve(float const* const input,
           float* const output,
           int const N) {  
    if (N <= maxInputsPerSolveCall) {
        float* buffer;
        gpuErrchk(hipMalloc((void**)&buffer, ((N + 1) / 2) * sizeof(float)));
        dim3 const threadsPerBlock = dim3(1024);
        // There should always be only 1 block.
        dim3 const blocksPerGrid = dim3(CEIL_DIV(N, (2 * threadsPerBlock.x)));
        sum_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, N, buffer);
    } else {
        unsigned int const numSolveCalls = CEIL_DIV(N, maxInputsPerSolveCall); 
        float* solveCallOutputs;
        gpuErrchk(hipMalloc((void**)&solveCallOutputs, numSolveCalls * sizeof(float)));

        dim3 const threadsPerBlock = dim3(1024);
        dim3 const blocksPerGrid = dim3(1);
        for (int numProcessedItems = 0; numProcessedItems < N; numProcessedItems += maxInputsPerSolveCall) {
            unsigned int callNumber = numProcessedItems / maxInputsPerSolveCall;
            unsigned int const numItemsForCall = min(maxInputsPerSolveCall, N - numProcessedItems);
            float* buffer;
            gpuErrchk(hipMalloc((void**)&buffer, ((numItemsForCall + 1) / 2) * sizeof(float)));
            sum_kernel<<<blocksPerGrid, threadsPerBlock>>>(input + numProcessedItems,
                                                           solveCallOutputs + callNumber,
                                                           numItemsForCall,
                                                           buffer);
            gpuErrchk(hipFree(buffer));
        }
        solve(solveCallOutputs, output, numSolveCalls);
        gpuErrchk(hipFree(solveCallOutputs));
    }
}
