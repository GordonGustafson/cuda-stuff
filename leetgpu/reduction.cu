#include "hip/hip_runtime.h"
#include "solve.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define CEIL_DIV(dividend, divisor) (((dividend) + (divisor) - 1) / (divisor))

unsigned int const maxThreadsPerBlock = 1024;

__global__ void solve_kernel(float const* const input,
                             float* const output,
                             int const N) {  
    extern __shared__ float sharedBuffer[];
    sharedBuffer[threadIdx.x] = 0.0f;

    int const threadsPerGrid = gridDim.x * blockDim.x;
    int const firstElementIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (firstElementIdx < N) {
        int const secondElementIndex = firstElementIdx + threadsPerGrid;
        sharedBuffer[threadIdx.x] = input[firstElementIdx] + (secondElementIndex < N ? input[secondElementIndex] : 0);
    }

    for (int numActiveThreads = blockDim.x / 2; numActiveThreads >= 1; numActiveThreads /= 2) {
        __syncthreads();
        if (threadIdx.x < numActiveThreads) {
            sharedBuffer[threadIdx.x] += sharedBuffer[threadIdx.x + numActiveThreads];
        }
    }

    if (threadIdx.x == 0) {
        atomicAdd(output, sharedBuffer[0]);
    }
}

// input, output are device pointers
void solve(const float* input, float* output, int N) {  
    int threadsPerBlock = maxThreadsPerBlock;
    int blocksPerGrid = CEIL_DIV(N, threadsPerBlock * 2);
    solve_kernel<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(float)>>>(input, output, N);
}
