#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>
#include <limits>
#include <stdio.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define CEIL_DIV(dividend, divisor) (((dividend) + (divisor) - 1) / (divisor))

#define ALL_THREADS_IN_WARP_MASK 0xffffffffu
#define THREADS_PER_WARP 32
#define WARPS_PER_BLOCK 32

// From https://stackoverflow.com/a/17401122
__device__ static float atomicMax(float* address, float val) {
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__global__ void max_kernel(float const* const input,
                           float* const maxBuffer,
                           int const N) {
    float __shared__ sharedBuffer[WARPS_PER_BLOCK];
    int const i = blockIdx.x * blockDim.x + threadIdx.x;
    float localMax = (i < N) ? input[i] : -INFINITY;

    for (int numActiveThreadsInWarp = THREADS_PER_WARP / 2; numActiveThreadsInWarp >= 1; numActiveThreadsInWarp /= 2) {
        localMax = ::fmax(localMax, __shfl_down_sync(ALL_THREADS_IN_WARP_MASK, localMax, numActiveThreadsInWarp));
    }
    int const warpIdx = threadIdx.x / THREADS_PER_WARP;
    int const lane = threadIdx.x % THREADS_PER_WARP;

    if (lane == 0) {
        sharedBuffer[warpIdx] = localMax;
    }
    __syncthreads();

    if (threadIdx.x < THREADS_PER_WARP) {
        localMax = sharedBuffer[lane];

        for (int numActiveThreads = THREADS_PER_WARP / 2; numActiveThreads >= 1; numActiveThreads /= 2) {
            localMax = ::fmax(localMax, __shfl_down_sync(ALL_THREADS_IN_WARP_MASK, localMax, numActiveThreads));
        }
        if (threadIdx.x == 0) {
            atomicMax(maxBuffer, localMax);
        }
    }
}

__global__ void sum_exp_minus_max_kernel(float const* const input,
                                         float* const maxBuffer,
                                         float* const sumBuffer,
                                         int const N) {
    float __shared__ sharedBuffer[WARPS_PER_BLOCK];
    int const i = blockIdx.x * blockDim.x + threadIdx.x;
    float localSum = (i < N) ? expf(input[i] - *maxBuffer) : 0.0f;

    for (int numActiveThreads = THREADS_PER_WARP / 2; numActiveThreads >= 1; numActiveThreads /= 2) {
        localSum += __shfl_down_sync(ALL_THREADS_IN_WARP_MASK, localSum, numActiveThreads);
    }

    if (threadIdx.x % THREADS_PER_WARP == 0) {
        sharedBuffer[threadIdx.x / THREADS_PER_WARP] = localSum;
    }

    __syncthreads();

    if (threadIdx.x < WARPS_PER_BLOCK) {
        localSum = sharedBuffer[threadIdx.x];

        for (int numActiveThreads = THREADS_PER_WARP / 2; numActiveThreads >= 1; numActiveThreads /= 2) {
            localSum += __shfl_down_sync(ALL_THREADS_IN_WARP_MASK, localSum, numActiveThreads);
        }

        if (threadIdx.x == 0) {
            atomicAdd(sumBuffer, localSum);
        }
    }
}

__global__ void softmax_kernel(float const* const input,
                               float* const output,
                               float* const maxBuffer,
                               float* const sumBuffer,
                               int const N) {
    int const i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        float const x_i = input[i];
        float const exp_x_i_minus_max = expf(x_i - *maxBuffer);
        output[i] = exp_x_i_minus_max / *sumBuffer;
    }
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* input, float* output, int N) {

    float zeroArray[1] = {0.0f};
    float negInfArray[1] = {-INFINITY};

    float* maxBuffer_d;
    gpuErrchk(hipMalloc((void**)&maxBuffer_d, 1 * sizeof(float)));
    gpuErrchk(hipMemcpy(maxBuffer_d, negInfArray, 1 * sizeof(float), hipMemcpyHostToDevice));

    float* sumBuffer_d;
    gpuErrchk(hipMalloc((void**)&sumBuffer_d, 1 * sizeof(float)));
    gpuErrchk(hipMemcpy(sumBuffer_d, zeroArray, 1 * sizeof(float), hipMemcpyHostToDevice));

    int threadsPerBlock = 1024;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    max_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, maxBuffer_d, N);
    sum_exp_minus_max_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, maxBuffer_d, sumBuffer_d, N);
    softmax_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, maxBuffer_d, sumBuffer_d, N);
    hipDeviceSynchronize();
}
