#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void matrix_multiplication_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    int const row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < M) {
        float result = 0.0f;
        for (int col = 0; col < K; col++) {
            for (int i = 0; i < N; i++) {
                result += A[row * N + i] * B[i * K + col];
            }
            C[row * K + col] = result;
        }
    }
}

// A, B, C are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* A, const float* B, float* C, int M, int N, int K) {
    dim3 threadsPerBlock(32, 1);
    dim3 blocksPerGrid((M + threadsPerBlock.x - 1) / threadsPerBlock.x, 1)
    
    matrix_multiplication_kernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();
}
