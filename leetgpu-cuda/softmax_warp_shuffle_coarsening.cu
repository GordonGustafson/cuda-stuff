#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>
#include <limits>
#include <stdio.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define CEIL_DIV(dividend, divisor) (((dividend) + (divisor) - 1) / (divisor))

#define ALL_THREADS_IN_WARP_MASK 0xffffffffu
#define THREADS_PER_WARP 32
#define WARPS_PER_BLOCK 32

#define COARSENING_FACTOR 13

// From https://stackoverflow.com/a/17401122
__device__ static float atomicMax(float* address, float val) {
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__global__ void max_kernel(float const* const input,
                           float* const maxBuffer,
                           int const N) {
    float __shared__ sharedBuffer[WARPS_PER_BLOCK];
    int const index = blockIdx.x * blockDim.x + threadIdx.x;
    float localMax = (index < N) ? input[index] : -INFINITY;

    int const threadsPerGrid = gridDim.x * blockDim.x;
    for (int nextElementIndex = index + threadsPerGrid; nextElementIndex < N; nextElementIndex += threadsPerGrid) {
        localMax = ::fmax(localMax, input[nextElementIndex]);
    }

    for (int numActiveThreadsInWarp = THREADS_PER_WARP / 2; numActiveThreadsInWarp >= 1; numActiveThreadsInWarp /= 2) {
        localMax = ::fmax(localMax, __shfl_down_sync(ALL_THREADS_IN_WARP_MASK, localMax, numActiveThreadsInWarp));
    }
    int const warpIdx = threadIdx.x / THREADS_PER_WARP;
    int const lane = threadIdx.x % THREADS_PER_WARP;

    if (lane == 0) {
        sharedBuffer[warpIdx] = localMax;
    }
    __syncthreads();

    if (threadIdx.x < THREADS_PER_WARP) {
        localMax = sharedBuffer[lane];

        for (int numActiveThreads = THREADS_PER_WARP / 2; numActiveThreads >= 1; numActiveThreads /= 2) {
            localMax = ::fmax(localMax, __shfl_down_sync(ALL_THREADS_IN_WARP_MASK, localMax, numActiveThreads));
        }
        if (threadIdx.x == 0) {
            atomicMax(maxBuffer, localMax);
        }
    }
}

__global__ void sum_exp_minus_max_kernel(float const* const input,
                                         float* const maxBuffer,
                                         float* const sumBuffer,
                                         int const N) {
    float __shared__ sharedBuffer[WARPS_PER_BLOCK];
    int const index = blockIdx.x * blockDim.x + threadIdx.x;
    float const globalMax = *maxBuffer;
    float localSum = (index < N) ? expf(input[index] - globalMax) : 0.0f;


    int const threadsPerGrid = gridDim.x * blockDim.x;
    for (int nextElementIndex = index + threadsPerGrid; nextElementIndex < N; nextElementIndex += threadsPerGrid) {
        localSum += expf(input[nextElementIndex] - globalMax);
    }

    for (int numActiveThreads = THREADS_PER_WARP / 2; numActiveThreads >= 1; numActiveThreads /= 2) {
        localSum += __shfl_down_sync(ALL_THREADS_IN_WARP_MASK, localSum, numActiveThreads);
    }

    if (threadIdx.x % THREADS_PER_WARP == 0) {
        sharedBuffer[threadIdx.x / THREADS_PER_WARP] = localSum;
    }

    __syncthreads();

    if (threadIdx.x < WARPS_PER_BLOCK) {
        localSum = sharedBuffer[threadIdx.x];

        for (int numActiveThreads = THREADS_PER_WARP / 2; numActiveThreads >= 1; numActiveThreads /= 2) {
            localSum += __shfl_down_sync(ALL_THREADS_IN_WARP_MASK, localSum, numActiveThreads);
        }

        if (threadIdx.x == 0) {
            atomicAdd(sumBuffer, localSum);
        }
    }
}

__global__ void softmax_kernel(float const* const input,
                               float* const output,
                               float* const maxBuffer,
                               float* const sumBuffer,
                               int const N) {
    float const globalMax = *maxBuffer;
    float const globalSum = *sumBuffer;

    int const index = blockIdx.x * blockDim.x + threadIdx.x;
    int const threadsPerGrid = gridDim.x * blockDim.x;
    for (int nextElementIndex = index; nextElementIndex < N; nextElementIndex += threadsPerGrid) {
        float const x_i = input[nextElementIndex];
        float const exp_x_i_minus_max = expf(x_i - globalMax);
        output[nextElementIndex] = exp_x_i_minus_max / globalSum;
    }
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* input, float* output, int N) {

    float zeroArray[1] = {0.0f};
    float negInfArray[1] = {-INFINITY};

    float* maxBuffer_d;
    gpuErrchk(hipMalloc((void**)&maxBuffer_d, 1 * sizeof(float)));
    gpuErrchk(hipMemcpy(maxBuffer_d, negInfArray, 1 * sizeof(float), hipMemcpyHostToDevice));

    float* sumBuffer_d;
    gpuErrchk(hipMalloc((void**)&sumBuffer_d, 1 * sizeof(float)));
    gpuErrchk(hipMemcpy(sumBuffer_d, zeroArray, 1 * sizeof(float), hipMemcpyHostToDevice));

    int threadsPerBlock = 1024;
    int blocksPerGrid = CEIL_DIV(N, threadsPerBlock * COARSENING_FACTOR);
    max_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, maxBuffer_d, N);
    sum_exp_minus_max_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, maxBuffer_d, sumBuffer_d, N);
    softmax_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, maxBuffer_d, sumBuffer_d, N);
    hipDeviceSynchronize();
}

#define ARRAY_SIZE 500000

int main() {
    float* inputArray = new float[ARRAY_SIZE];
    for (int i = 0; i < ARRAY_SIZE; i++) {
        inputArray[i] = (float)i;
    }

    float* inputArray_d;
    gpuErrchk(hipMalloc((void**)&inputArray_d, ARRAY_SIZE * sizeof(float)));
    gpuErrchk(hipMemcpy(inputArray_d, inputArray, ARRAY_SIZE * sizeof(float), hipMemcpyHostToDevice));

    float* outputArray_d;
    gpuErrchk(hipMalloc((void**)&outputArray_d, ARRAY_SIZE * sizeof(float)));

    solve(inputArray_d, outputArray_d, ARRAY_SIZE);

    gpuErrchk(hipFree(inputArray_d));
    gpuErrchk(hipFree(outputArray_d));
    delete[] inputArray;

    return 0;
}
